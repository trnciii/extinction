#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "cuda.cuh"

#include "kernel.hpp"


__global__ void visibility(float* visible,
	const float* ray_slope, size_t steps,
	const float* height, size_t realizations, size_t length)
{
	size_t s = blockIdx.x*blockDim.x + threadIdx.x;
	if(s>=steps) return;
	visible[s] = g1_distant(height, realizations, length, ray_slope[s]);
}

void call_visibility(float* visible,
	const float* ray_slope, size_t steps,
	const float* height, size_t realizations, size_t length)
{
	float* d_visible;
	hipMallocManaged(&d_visible, steps*sizeof(float));

	float* d_ray_slope;
	hipMallocManaged(&d_ray_slope, steps*sizeof(float));
	hipMemcpy(d_ray_slope, ray_slope, steps*sizeof(float), hipMemcpyDefault);

	float* d_height;
	hipMallocManaged(&d_height, realizations*length*sizeof(float));
	hipMemcpy(d_height, height, realizations*length*sizeof(float), hipMemcpyDefault);

	int threads = 1024;
	visibility <<< steps/threads + 1, threads >>> (d_visible, d_ray_slope, steps, d_height, realizations, length);
	hipDeviceSynchronize();

	hipMemcpy(visible, d_visible, steps*sizeof(float), hipMemcpyDefault);

	hipFree(d_visible);
	hipFree(d_ray_slope);
	hipFree(d_height);
}