#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "kernel.cuh"

__global__ void visibility(float* x, size_t n){
	size_t i = blockIdx.x*blockDim.x + threadIdx.x;

	if(i>=n) return;

	x[i] += 10;
}

void call_visibility(float* x, size_t n){
	float* y;
	size_t size = n*sizeof(float);
	hipMallocManaged(&y, size);
	hipMemcpy(y, x, size, hipMemcpyDefault);

	int threads = 1024;
	visibility <<<n/threads + 1, threads>>> (y, n);
	hipDeviceSynchronize();

	hipMemcpy(x, y, size, hipMemcpyDefault);
	hipFree(y);
}