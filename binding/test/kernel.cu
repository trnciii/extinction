#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "kernel.cuh"

__global__ void visibility(float* visible,
	float* slope, size_t steps,
	float* height, size_t realizations, size_t length)
{
	size_t s = blockIdx.x*blockDim.x + threadIdx.x;
	if(s>=steps) return;

	visible[s] = realizations;
	for(size_t m=0; m<realizations; m++){
		for(size_t i=0; i<length; i++){
			if(height[m*length + 0] + slope[s]*i < height[m*length + i]){
				visible[s] -= 1;
				break;
			}
		}
	}

	visible[s] /= realizations;
}

void call_visibility(float* visible,
	float* slope, size_t steps,
	float* height, size_t realizations, size_t length)
{
	float* d_visible;
	hipMallocManaged(&d_visible, steps*sizeof(float));

	float* d_slope;
	hipMallocManaged(&d_slope, steps*sizeof(float));
	hipMemcpy(d_slope, slope, steps*sizeof(float), hipMemcpyDefault);

	float* d_height;
	hipMallocManaged(&d_height, realizations*length*sizeof(float));
	hipMemcpy(d_height, height, realizations*length*sizeof(float), hipMemcpyDefault);

	int threads = 1024;
	visibility <<< steps/threads + 1, threads >>> (d_visible, d_slope, steps, d_height, realizations, length);
	hipDeviceSynchronize();

	hipMemcpy(visible, d_visible, steps*sizeof(float), hipMemcpyDefault);

	hipFree(d_visible);
	hipFree(d_slope);
	hipFree(d_height);
}