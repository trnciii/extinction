#include "hip/hip_runtime.h"
#include "kernel.h"

#include <hip/hip_runtime.h> 
#include "vector_math.h"


__global__ void kernel(float* x, uint32_t n){
	uint32_t i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i>=n) return;
	x[i] += 10;
}


extern "C" void call_kernel(float* x, uint32_t n){
	for(int i=0; i<n; i++){
		std::cout <<x[i] <<", ";
	}
	std::cout <<std::endl;

	float* y;
	size_t size = n*sizeof(float);
	hipMallocManaged(&y, size);
	hipMemcpy(y, x, size, hipMemcpyDefault);

	kernel <<<n/1024 + 1, 1024>>> (y, n);
	hipDeviceSynchronize();

	hipMemcpy(x, y, size, hipMemcpyDefault);
	for(int i=0; i<n; i++){
		std::cout <<x[i] <<", "
;	}
	std::cout <<std::endl;
}
